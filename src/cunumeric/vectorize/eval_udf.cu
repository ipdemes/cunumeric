/* Copyright 2024 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/vectorize/eval_udf.h"
#include "cunumeric/vectorize/eval_udf_template.inl"
#include "cunumeric/execution_policy/indexing/parallel_loop.cuh"

namespace cunumeric {

/*static*/ void EvalUdfTask::gpu_variant(TaskContext& context)
{
  eval_udf_template<VariantKind::GPU>(context);
}

template <>
struct UDF<VariantKind::GPU> {
  hipFunction_t udf;
  UDF() {}
  UDF(int64_t hash, uint64_t) { udf = get_udf(hash); }
  __CUDA_HD__ void call_udf_dense(const size_t idx) const
  {
    printf("IRINA DEBUG inside GPU kernel");
  }
  template <int DIM = 1>
  __CUDA_HD__ void call_udf_sparse(const size_t idx, Point<DIM>& p) const
  {
    printf("IRINA DEBUG inside GPU kernel");
  }
};

}  // namespace cunumeric
