#include "hip/hip_runtime.h"
/* Copyright 2023 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/vectorize/eval_udf.h"
#include "cunumeric/cuda_help.h"
#include <regex>
#include <hip/hip_runtime.h>

namespace cunumeric {

using namespace Legion;
using namespace legate;

struct EvalUdfGPU {
  template <LegateTypeCode CODE, int DIM>
  void operator()(EvalUdfArgs& args) const
  {
    using VAL = legate_type_of<CODE>;
    //auto rect = args.inputs[0].shape<DIM>();
    //if (rect.empty()) return;

    // 1: we need to vreate a function from the ptx generated y numba
    const unsigned num_options   = 4;
    const size_t log_buffer_size = 16384;
    std::vector<char> log_info_buffer(log_buffer_size);
    std::vector<char> log_error_buffer(log_buffer_size);
    hipJitOption jit_options[] = {
      hipJitOptionInfoLogBuffer,
      hipJitOptionInfoLogBufferSizeBytes,
      hipJitOptionErrorLogBuffer,
      hipJitOptionErrorLogBufferSizeBytes,
    };
    void* option_vals[] = {
      static_cast<void*>(log_info_buffer.data()),
      reinterpret_cast<void*>(log_buffer_size),
      static_cast<void*>(log_error_buffer.data()),
      reinterpret_cast<void*>(log_buffer_size),
    };

    hipModule_t module;
    hipError_t result =
      hipModuleLoadDataEx(&module, args.ptx.data(), num_options, jit_options, option_vals);
    if (result != hipSuccess) {
      if (result == hipErrorOperatingSystem) {
        fprintf(stderr,
                "ERROR: Device side asserts are not supported by the "
                "CUDA driver for MAC OSX, see NVBugs 1628896.\n");
        exit(-1);
      } else if (result == hipErrorNoBinaryForGpu) {
        fprintf(stderr, "ERROR: The binary was compiled for the wrong GPU architecture.\n");
        exit(-1);
      } else {
        fprintf(stderr, "Failed to load CUDA module! Error log: %s\n", log_error_buffer.data());
#if CUDA_VERSION >= 6050
        const char *name, *str;
        assert(hipDrvGetErrorName(result, &name) == hipSuccess);
        assert(hipDrvGetErrorString(result, &str) == hipSuccess);
        fprintf(stderr, "CU: hipModuleLoadDataEx = %d (%s): %s\n", result, name, str);
#else
        fprintf(stderr, "CU: hipModuleLoadDataEx = %d\n", result);
#endif
        exit(-1);
      }
    }

    std::cmatch line_match;
    bool match =
      std::regex_search(args.ptx.data(), line_match, std::regex(".visible .entry [_a-zA-Z0-9$]+"));
#ifdef DEBUG_CUNUMERIC
    assert(match);
#endif
    const auto& matched_line = line_match.begin()->str();
    auto fun_name = matched_line.substr(matched_line.rfind(" ") + 1, matched_line.size());

    hipFunction_t func;
    result = hipModuleGetFunction(&func, module, fun_name.c_str());
#ifdef DEBUG_CUNUMERIC
    assert(result == hipSuccess);
#endif

    // 2: after fucntion is generated, we can execute it:

    // Filling up the buffer with arguments
    size_t buffer_size = (args.inputs.size()+args.scalars.size()) * sizeof(void*);
    buffer_size += sizeof(size_t);

    std::vector<char> arg_buffer(buffer_size);
    char* raw_arg_buffer = arg_buffer.data();

    auto p = raw_arg_buffer;

    size_t size =1;
    if (args.inputs.size()>0){
      auto rect = args.inputs[0].shape<DIM>();
      size = rect.volume();
      for (size_t i = 0; i < args.inputs.size(); i++) {
        if (i < args.num_outputs) {
          auto out                           = args.outputs[i].write_accessor<VAL, DIM>(rect);
          *reinterpret_cast<const void**>(p) = out.ptr(rect);
        } else {
          auto in                            = args.inputs[i].read_accessor<VAL, DIM>(rect);
          *reinterpret_cast<const void**>(p) = in.ptr(rect);
        }
        p += sizeof(void*);
      }
    }
    for (auto scalar: args.scalars){
        memcpy(p, scalar.ptr(), scalar.size());
        p += scalar.size();
       // *reinterpret_cast<const void**>(p) =s;
        //p += sizeof(void*);
      }
    memcpy(p, &size, sizeof(size_t));

    void* config[] = {
      HIP_LAUNCH_PARAM_BUFFER_POINTER,
      static_cast<void*>(raw_arg_buffer),
      HIP_LAUNCH_PARAM_BUFFER_SIZE,
      &buffer_size,
      HIP_LAUNCH_PARAM_END,
    };

    const uint32_t gridDimX = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    const uint32_t gridDimY = 1;
    const uint32_t gridDimZ = 1;

    const uint32_t blockDimX = THREADS_PER_BLOCK;
    const uint32_t blockDimY = 1;
    const uint32_t blockDimZ = 1;

    auto stream = get_cached_stream();

    // executing the function
    hipError_t status = hipModuleLaunchKernel(
      func, gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, 0, stream, NULL, config);
    if (status != hipSuccess) {
      fprintf(stderr, "Failed to launch a CUDA kernel\n");
      exit(-1);
    }

    CHECK_CUDA_STREAM(stream);
  }
};

/*static*/ void EvalUdfTask::gpu_variant(TaskContext& context)
{
  std::vector<Scalar>scalars;
  for (size_t i=2; i<context.scalars().size(); i++)
      scalars.push_back(context.scalars()[i]);

  EvalUdfArgs args{0,
                   context.inputs(),
                   context.outputs(),
                   scalars,
                   context.scalars()[0].value<std::string>(),
                   context.scalars()[1].value<uint32_t>()};
  size_t dim=1;
  if (args.inputs.size()>0){
    dim = args.inputs[0].dim() == 0 ? 1 : args.inputs[0].dim();
    double_dispatch(dim, args.inputs[0].code(), EvalUdfGPU{}, args);
  }
  else{
    double_dispatch(dim, args.inputs[0].code(), EvalUdfGPU{}, args);
    //double_dispatch(dim, 0 , EvalUdfGPU{}, args);
  }
}
}  // namespace cunumeric
